#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "perceptron.h"

// Softmax activation function implemented in device code
__device__ void softmax(double* input, double* output, int length) {
    double max_input = input[0];
    for (int i = 1; i < length; i++) {
        if (input[i] > max_input) {
            max_input = input[i];
        }
    }

    double sum = 0.0;
    for (int i = 0; i < length; i++) {
        output[i] = exp(input[i] - max_input);
        sum += output[i];
    }

    for (int i = 0; i < length; i++) {
        output[i] /= sum;
    }
}

// CUDA kernel to train the perceptron
__global__ void train_perceptron_kernel(double *weights, double *biases, double *inputs, double *labels, int input_size, int num_classes, int num_samples, double learning_rate) {
    int sample = blockIdx.x * blockDim.x + threadIdx.x;
    if (sample < num_samples) {
        double *input = &inputs[sample * input_size];
        double *label = &labels[sample * num_classes];

        double weighted_sum[num_classes];
        double output[num_classes];

        for (int j = 0; j < num_classes; j++) {
            weighted_sum[j] = 0.0;
            for (int i = 0; i < input_size; i++) {
                weighted_sum[j] += input[i] * weights[i * num_classes + j];
            }
            weighted_sum[j] += biases[j];
        }

        softmax(weighted_sum, output, num_classes);

        for (int j = 0; j < num_classes; j++) {
            double error = label[j] - output[j];
            for (int i = 0; i < input_size; i++) {
                weights[i * num_classes + j] += learning_rate * error * input[i];
            }
            biases[j] += learning_rate * error;
        }
    }
}

// Host function to train perceptron using CUDA
void train_perceptron_cuda(Perceptron* p, double **inputs, double **labels, int num_samples, int epochs, double learning_rate) {
    double *d_weights, *d_biases, *d_inputs, *d_labels;

    size_t weights_size = p->input_size * p->num_classes * sizeof(double);
    size_t biases_size = p->num_classes * sizeof(double);
    size_t inputs_size = num_samples * p->input_size * sizeof(double);
    size_t labels_size = num_samples * p->num_classes * sizeof(double);

    hipMalloc(&d_weights, weights_switch);
    hipMalloc(&d_biases, biases_size);
    hipMalloc(&d_inputs, inputs_size);
    hipMalloc(&d_labels, labels_size);

    hipMemcpy(d_weights, p->weights, weights_size, hipMemcpyHostToDevice);
    hipMemcpy(d_biases, p->biases, biases_size, hipMemcpyHostToDevice);
    hipMemcpy(d_inputs, *inputs, inputs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, *labels, labels_size, hipMemcpyHostToDevice);

    int block_size = 256; // Adjust based on your GPU capabilities (mine is Nvidia RTX 3060 Mobile/Max-Q) 
    int num_blocks = (num_samples + block_size - 1) / block_size;

    for (int epoch = 0; epoch < epochs; epoch++) {
        train_perceptron_kernel<<<num_blocks, block_size>>>(d_weights, d_biases, d_inputs, d_labels, p->input_size, p->num_classes, num_samples, learning_rate);
        hipDeviceSynchronize();
    }

    hipMemcpy(p->weights, d_weights, weights_size, hipMemcpyDeviceToHost);
    hipMemcpy(p->biases, d_biases, biases_size, hipMemcpyDeviceToHost);

    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_inputs);
    hipFree(d_labels);
}

